
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorAddition(float *v1, float *v2, float *o, int length) {
  int i = threadIdx.x;
  if (i < length) {
    o[i] = v1[i] + v2[i];
  }
}

int main() {
  int N = 10;
  float *v1 = (float*) malloc(N*sizeof(float));
  float *v2 = (float*) malloc(N*sizeof(float));
  float *o = (float*) malloc(N*sizeof(float));

  for (int i = 0; i < N; i++) {
    v1[i] = 1;
    v2[i] = 1;
  }  

  float* v1Gpu;
  hipError_t err1 = hipMalloc(&v1Gpu, N*sizeof(float));
  float* v2Gpu;
  hipError_t err2 = hipMalloc(&v2Gpu, N*sizeof(float));
  float* oGpu;
  hipError_t err3 = hipMalloc(&oGpu, N*sizeof(float));

  hipMemcpy(v1Gpu, v1, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(v2Gpu, v2, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(oGpu, o, N*sizeof(float), hipMemcpyHostToDevice);

  vectorAddition<<<1, N>>>(v1Gpu, v2Gpu, oGpu, N);

  hipMemcpy(o, oGpu, N*sizeof(float), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++) {
    printf("%f ", v1[i]);
  }
  printf("\n"); 
  for (int i = 0; i < N; i++) {
    printf("%f ", v2[i]);
  }
  printf("\n"); 
  for (int i = 0; i < N; i++) {
    printf("%f ", o[i]);
  }
  printf("\n"); 

  hipFree(v1Gpu);
  hipFree(v2Gpu);
  hipFree(oGpu);
  free(v1);
  free(v2);
  free(o);
}